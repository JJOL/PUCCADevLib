#include "hip/hip_runtime.h"
#include "GoLCCA.cuh"
#include <stdlib.h>
#include <string.h>
#include ""
#include "mat_utils.h"

void hostClean(void* address) { if (address != NULL) free(address); }
void deviceClean(void* address) { if (address != NULL) hipFree(address); }

GoLCCA::GoLCCA(int gridN) : m_gridN(gridN)
{
	env = ENV_UNSPECIFIED;
	caGlobal.gridN = gridN;
	// Init to NULL to be able to skip them during cleaning if we have no data
	caGlobal.neighborsKernel = NULL;
	m_nextData = NULL;
	dev_prevData = NULL;
	dev_nextData = NULL;
	dev_ca = NULL;
	m_kernel = NULL;
	m_prevData = NULL;
	m_data = NULL;
}

void GoLCCA::init(ExecutionEnv _env)
{
	env = _env;

	const int size = m_gridN * m_gridN;

	m_data = (int*)malloc(size * sizeof(int));
	initMat(m_data, m_gridN, 0);

	// Setup 
	int x, y;
	m_prevData = new GoLCell[size];
	for (x = 0; x < m_gridN; x++) {
		for (y = 0; y < m_gridN; y++) {
			m_prevData[caGlobal.f2dTo1d(x, y)].x = x;
			m_prevData[caGlobal.f2dTo1d(x, y)].y = y;
		}
	}
		
	m_nextData = new GoLCell[size];
	
	// Only GoL data
	caGlobal.neighborsKernel = (int*)malloc(3 * 3 * sizeof(int));
	initMooreKernel(caGlobal.neighborsKernel);

	if (env == ENV_GPU) {
		// Creating memory pointers for state on device
		hipMalloc((void**)&dev_prevData, size * sizeof(GoLCell));
		hipMalloc((void**)&dev_nextData, size * sizeof(GoLCell));

		hipMemcpy(dev_prevData, m_prevData, size * sizeof(GoLCell), hipMemcpyHostToDevice);
		hipMemcpy(dev_nextData, m_nextData, size * sizeof(GoLCell), hipMemcpyHostToDevice);

		// Setting Up Common GoL device auxilary data
		GoLGlobals cpyGlobals;
		cpyGlobals.gridN = caGlobal.gridN;
		hipMalloc((void**)&cpyGlobals.neighborsKernel, 3 * 3 * sizeof(int));
		hipMalloc((void**)&dev_caGlobals, sizeof(GoLGlobals));
		hipMemcpy(cpyGlobals.neighborsKernel, caGlobal.neighborsKernel, 3 * 3 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_caGlobals, &cpyGlobals, sizeof(GoLGlobals), hipMemcpyHostToDevice);
	}
}

GoLCCA::~GoLCCA()
{
	hostClean(caGlobal.neighborsKernel);
	hostClean(m_data);
	hostClean(m_prevData);
	hostClean(m_nextData);

	if (env == ENV_GPU) {
		deviceClean(dev_prevData);
		deviceClean(dev_nextData);
		GoLGlobals cpyDevGlobals;
		hipMemcpy(&cpyDevGlobals, dev_caGlobals, sizeof(GoLGlobals), hipMemcpyDeviceToHost);
		deviceClean(cpyDevGlobals.neighborsKernel);
		deviceClean(dev_caGlobals);
	}
}

void GoLCCA::setState(void* stateData)
{
	memcpy(m_prevData, stateData, m_gridN * m_gridN * sizeof(GoLCell));
}

void* GoLCCA::getState()
{
	return m_prevData;
}

void GoLCCA::prepare()
{
	int size = m_gridN * m_gridN;

	memcpy(m_nextData, m_prevData, m_gridN * m_gridN * sizeof(GoLCell));

	if (env == ENV_GPU) {
		hipMemcpy(dev_prevData, m_prevData, size * sizeof(GoLCell), hipMemcpyHostToDevice);
		hipMemcpy(dev_nextData, m_nextData, size * sizeof(GoLCell), hipMemcpyHostToDevice);
	}
}

int* GoLCCA::getStateIntMat()
{
	for (int i = 0; i < m_gridN * m_gridN; i++)
		m_data[i] = m_prevData[i].state;
	return m_data;
}

void GoLCCA::cpuStep()
{
	GoLCell* temp;
	for (int y = 1; y < m_gridN - 1; y++) {
		for (int x = 1; x < m_gridN - 1; x++) {
			m_nextData[caGlobal.f2dTo1d(x, y)].update(&caGlobal, m_prevData);
		}
	}

	temp = m_prevData;
	m_prevData = m_nextData;
	m_nextData = temp;
}

__global__ void kGoLStep(GoLGlobals* ca, GoLCell* oldCAData, GoLCell* newCAData, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < n) {
		newCAData[tid].update(ca, oldCAData);
		tid += blockDim.x + gridDim.x;
	}

}

void GoLCCA::gpuStep()
{
	int size = m_gridN * m_gridN;

	GoLCell* dev_temp;

	if (dev_caGlobals == NULL) {
		fprintf(stderr, "ERROR: GoL dev_caGlobals is empty!\n");
		return;
	}
	kGoLStep <<<(size + size - 1) / 32, 32>> > (dev_caGlobals, dev_prevData, dev_nextData, size);
	hipDeviceSynchronize();
	dev_temp = dev_prevData;
	dev_prevData = dev_nextData;
	dev_nextData = dev_temp;

	hipMemcpy(m_prevData, dev_prevData, size * sizeof(GoLCell), hipMemcpyDeviceToHost);
}



__host__ __device__ void GoLGlobals::f1dTo2d(int index, int* x, int* y)
{
	*x = index % gridN;
	*y = index / gridN;
}
 __forceinline__ __host__ __device__ int GoLGlobals::f2dTo1d(int x, int y)
{
	return x + y * gridN;
}



/* GoL Closest Logic at Cell Level */
__host__ __device__ void GoLCell::update(GoLGlobals* ca, GoLCell* prevCAData)
{
	int aliveNeighbors = 0;

	for (int k_row = 0; k_row < 3; k_row++) {
		for (int k_col = 0; k_col < 3; k_col++) {
			aliveNeighbors +=
				ca->neighborsKernel
				[k_col + (k_row * 3)] * prevCAData[ca->f2dTo1d(x - 1 + k_col, y - 1 + k_row)].state;
		}
	}

	state = (1 - state) * (aliveNeighbors == 3) + (state) * (aliveNeighbors == 2 || aliveNeighbors == 3);
}